#include "hip/hip_runtime.h"
#include "Math_CUDA.h"

template <class DataType>
__global__ void vector_sum_kernel(llint_t N, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<N)
	{
		in1[i] += in2[i];
	}
}

template <class DataType>
void vector_sum(llint_t N, DataType* in1, DataType* in2)
{
	int threads = 512;
	int blocks = N/512+1;
	vector_sum_kernel<<<blocks,threads>>>(N,in1,in2);
}

template <class DataType>
__global__ void vector_product_kernel(llint_t N, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<N)
	{
		in1[i] *= in2[i];
	}
}

template <class DataType>
void vector_product(llint_t N, DataType* in1, DataType* in2)
{
	int threads = 512;
	int blocks = N/512+1;
	vector_product_kernel<<<blocks,threads>>>(N,in1,in2);
}

template <class DataType>
__global__ void vector_diff_kernel(llint_t N, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<N)
	{
		in1[i] -= in2[i];
	}
}

template <class DataType>
void vector_diff(llint_t N, DataType* in1, DataType* in2)
{
	int threads = 512;
	int blocks = N/512+1;
	vector_diff_kernel<<<blocks,threads>>>(N,in1,in2);
}

template <class DataType>
__global__ void vector_div_kernel(llint_t N, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<N)
	{
		in1[i] /= in2[i];
	}
}

template <class DataType>
void vector_div(llint_t N, DataType* in1, DataType* in2)
{
	int threads = 512;
	int blocks = N/512+1;
	vector_div_kernel<<<blocks,threads>>>(N,in1,in2);
}

template <class DataType>
__global__ void matrix_sum_kernel(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	long long int j = blockIdx.y*blockDim.y+threadIdx.y;
	if(i<Nr && j<Nc)
	{
		in1[i*Nc+j] += in2[i*Nc+j];
	}
}

template <class DataType>
void martrix_sum(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	dim3 threads(512, 512);
    dim3 blocks(Nr/512+1, Nc/512+1);
	matrix_sum_kernel<<<blocks,threads>>>(Nr,Nc,in1,in2);
}

template <class DataType>
__global__ void matrix_prod_kernel(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	long long int j = blockIdx.y*blockDim.y+threadIdx.y;
	if(i<Nr && j<Nc)
	{
		in1[i*Nc+j] *= in2[i*Nc+j];
	}
}

template <class DataType>
void martrix_prod(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	dim3 threads(512, 512);
    dim3 blocks(Nr/512+1, Nc/512+1);
	matrix_prod_kernel<<<blocks,threads>>>(Nr,Nc,in1,in2);
}

template <class DataType>
__global__ void matrix_diff_kernel(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	long long int j = blockIdx.y*blockDim.y+threadIdx.y;
	if(i<Nr && j<Nc)
	{
		in1[i*Nc+j] -= in2[i*Nc+j];
	}
}

template <class DataType>
void martrix_diff(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	dim3 threads(512, 512);
    dim3 blocks(Nr/512+1, Nc/512+1);
	matrix_diff_kernel<<<blocks,threads>>>(Nr,Nc,in1,in2);
}

template <class DataType>
__global__ void matrix_div_kernel(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	long long int i = blockIdx.x*blockDim.x+threadIdx.x;
	long long int j = blockIdx.y*blockDim.y+threadIdx.y;
	if(i<Nr && j<Nc)
	{
		in1[i*Nc+j] /= in2[i*Nc+j];
	}
}

template <class DataType>
void martrix_div(llint_t Nr, llint_t Nc, DataType* in1, DataType* in2)
{
	dim3 threads(512, 512);
    dim3 blocks(Nr/512+1, Nc/512+1);
	matrix_div_kernel<<<blocks,threads>>>(Nr,Nc,in1,in2);
}

template <class DataType>
__global__ void gradient_kernel(llint_t N, DataType* in, double* out, double h)
{
	double H1 = 1/h;
	double H2 = 1/2/h;
	
	if(i==0)
	{
		out[i] = (in[i+1]-in[i])*H1;
	}
	else if(i==(N-1))
	{
		out[i] = (in[i]-in[i-1])*H1;
	}
	else
	{
		in[i] = (in[i+1]-in[i-1])*H2;
	}
}

template <class DataType>
void gradient(llint_t N, DataType* in, double* out, double h)
{
	int threads = 512;
	int blocks = N/512+1;
	gradient_kernel<<<blocks,threads>>>(N,in,out,h);
}

template <class DataType>
__global__ void gradient_general_kernel(llint_t N, DataType* in, double* out, DataType* x)
{
	if(i==0)
	{
		out[i] = (in[i+1]-in[i])/(x[i+1]-x[i]);
	}
	else if(i==(N-1))
	{
		out[i] = (in[i]-in[i-1])/(x[i]-x[i-1]);
	}
	else
	{
		in[i] = (in[i+1]-in[i-1])/(x[i+1]-x[i-1]);
	}
}

template <class DataType>
void gradient_general(llint_t N, DataType* in, double* out, DataType* x)
{
	int threads = 512;
	int blocks = N/512+1;
	gradient_kernel<<<blocks,threads>>>(N,in,out,x);
}