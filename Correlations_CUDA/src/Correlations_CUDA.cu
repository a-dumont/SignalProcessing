#include "hip/hip_runtime.h"
#include "Correlations_CUDA.h"

__global__	void autocorrelation_cuda_kernel(long long int N, hipDoubleComplex* in, 
				double* out, int threads)
{
	// Compute the correlation
	long long int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = hipCreal(in[i])*hipCreal(in[i])+hipCimag(in[i])*hipCimag(in[i]);
	}
}

__global__	void autocorrelation_cuda_kernelf(long long int N, hipFloatComplex* in, 
				float* out, int threads)
{
	// Compute the correlation
	long long int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = hipCrealf(in[i])*hipCrealf(in[i])+hipCimagf(in[i])*hipCimagf(in[i]);
	}
}

template<class DataType>
void autocorrelation_cuda(long long int N, std::complex<DataType>* in, DataType* out){}

template<>
void autocorrelation_cuda<double>(long long int N, std::complex<double>* in, double* out)
{
	int threads = 512;
	long long int blocks = N/threads;
	autocorrelation_cuda_kernel<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipDoubleComplex*>(in),out,threads);
	hipDeviceSynchronize();
}

template<>
void autocorrelation_cuda<float>(long long int N, std::complex<float>* in, float* out)
{
	int threads = 512;
	long long int blocks = N/threads;
	autocorrelation_cuda_kernelf<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipFloatComplex*>(in),out,threads);
	hipDeviceSynchronize();
}

__global__	void cross_correlation_cuda_kernel(long long int N, hipDoubleComplex* in1, 
				hipDoubleComplex* in2, hipDoubleComplex* out, int threads)
{
	// Compute the correlation
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = hipCmul(in1[i],hipConj(in2[i]));
	}
}

__global__	void cross_correlation_cuda_kernelf(long long int N, hipFloatComplex* in1, 
				hipFloatComplex* in2, hipFloatComplex* out, int threads)
{
	// Compute the correlation
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = hipCmulf(in1[i],hipConjf(in2[i]));
	}
}

template<class DataType>
void cross_correlation_cuda(long long int N, std::complex<DataType>* in1, 
				std::complex<DataType>*in2, std::complex<DataType>* out){}

template<>
void cross_correlation_cuda<double>(long long int N, std::complex<double>* in1, 
				std::complex<double>* in2, std::complex<double>* out)
{
	int threads = 512;
	long long int blocks = N/threads;
	cross_correlation_cuda_kernel<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipDoubleComplex*>(in1),
					reinterpret_cast<hipDoubleComplex*>(in2),
					reinterpret_cast<hipDoubleComplex*>(out),threads);
	hipDeviceSynchronize();
}

template<>
void cross_correlation_cuda<float>(long long int N, std::complex<float>* in1, 
				std::complex<float>* in2, std::complex<float>* out)
{
	int threads = 512;
	long long int blocks = N/threads;
	cross_correlation_cuda_kernelf<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipFloatComplex*>(in1),
					reinterpret_cast<hipFloatComplex*>(in2),
					reinterpret_cast<hipFloatComplex*>(out),threads);
	hipDeviceSynchronize();
}

__global__	void complete_correlation_cuda_kernel(long long int N, hipDoubleComplex* in1, 
				hipDoubleComplex* in2, double* out1, double* out2, hipDoubleComplex* out3, 
				int threads)
{
	// Compute the correlation
	int i = threadIdx.x+blockIdx.x*threads;
	hipDoubleComplex a,b;
	if(i<N)
	{
		a = in1[i];
		b = in2[i];
		out1[i] = hipCreal(a)*hipCreal(a)+hipCimag(a)*hipCimag(a);
		out2[i] = hipCreal(b)*hipCreal(b)+hipCimag(b)*hipCimag(b);
		out3[i] = hipCmul(a,hipConj(b));
	}
}

__global__	void complete_correlation_cuda_kernelf(long long int N, hipFloatComplex* in1, 
				hipFloatComplex* in2, float* out1, float* out2, hipFloatComplex* out3, 
				int threads)
{
	// Compute the correlation
	int i = threadIdx.x+blockIdx.x*threads;
	hipFloatComplex a,b;
	if(i<N)
	{
		a = in1[i];
		b = in2[i];
		out1[i] = hipCrealf(a)*hipCrealf(a)+hipCimagf(a)*hipCimagf(a);
		out2[i] = hipCrealf(b)*hipCrealf(b)+hipCimagf(b)*hipCimagf(b);
		out3[i] = hipCmulf(a,hipConjf(b));
	}
}

template<class DataType>
void complete_correlation_cuda(long long int N, std::complex<DataType>* in1, 
				std::complex<DataType>*in2, 
				DataType* out1, 
				DataType* out2,
				std::complex<DataType>* out3){}

template<>
void complete_correlation_cuda<double>(long long int N, std::complex<double>* in1, 
				std::complex<double>* in2, double* out1, double* out2, std::complex<double>* out3)
{
	int threads = 512;
	long long int blocks = N/threads;
	complete_correlation_cuda_kernel<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipDoubleComplex*>(in1),
					reinterpret_cast<hipDoubleComplex*>(in2),
					out1,
					out2,
					reinterpret_cast<hipDoubleComplex*>(out3),threads);
	hipDeviceSynchronize();
}

template<>
void complete_correlation_cuda<float>(long long int N, std::complex<float>* in1, 
				std::complex<float>* in2, float* out1, float* out2, std::complex<float>* out3)
{
	int threads = 512;
	long long int blocks = N/threads;
	complete_correlation_cuda_kernelf<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipFloatComplex*>(in1),
					reinterpret_cast<hipFloatComplex*>(in2),
					out1,
					out2,
					reinterpret_cast<hipFloatComplex*>(out3),threads);
	hipDeviceSynchronize();
}

__global__ void reduction_kernel(long long int N, double* in, int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		in[i] += in[i+N];
	}
}

__global__ void reduction_kernelf(long long int N, float* in, int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		in[i] += in[i+N];
	}
}

template<class DataType> 
void reduction(long long int N, DataType* in, long long int size){}

template<>
void reduction<double>(long long int N, double* in, long long int size)
{
	if (N/2 >= size)
	{
		reduction_kernel<<<N/1024+1,512>>>(N/2,in,512);
		reduction(N/2,in,size);
	}
}

template<>
void reduction<float>(long long int N, float* in, long long int size)
{
	if (N/2 >= size)
	{
		reduction_kernelf<<<N/1024+1,512>>>(N/2,in,512);
		reduction(N/2,in,size);
	}
}


__global__ void reduction_general_kernel(long long int N, double* in, long long int size, 
				int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	long long int howmany = N/size;
	if(i<size)
	{
		for(long long int j=1;j<howmany;j++)
		{
			in[i] += in[i+j*size];
		}
	}
}

__global__ void reduction_general_kernelf(long long int N, float* in, long long int size, 
				int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	long long int howmany = N/size;
	if(i<size)
	{
		for(long long int j=1;j<howmany;j++)
		{
			in[i] += in[i+j*size];
		}
	}
}

template<class DataType> 
void reduction_general(long long int N, DataType* in, long long int size){}

template<>
void reduction_general<double>(long long int N, double* in, long long int size)
{
	int power = std::log2(N/size);
	long long int n = size*1<<power;
	if(n < N)
	{
		long long int diff = N-n+size;
		reduction_general_kernel<<<diff/512+1,512>>>(diff,in+n-size,size,512);
	}
	reduction(n,in,size);
}

template<>
void reduction_general<float>(long long int N, float* in, long long int size)
{
	int power = std::log2(N/size);
	long long int n = size*1<<power;
	if(n < N)
	{
		long long int diff = N-n+size;
		reduction_general_kernelf<<<diff/512+1,512>>>(diff,in+n-size,size,512);
	}
	reduction(n,in,size);
}

template<class DataType>
__global__ void convertComplex_kernel(long long int N, DataType* in, hipDoubleComplex* out,
				double conv, DataType offset, int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = make_hipDoubleComplex(conv*in[i]-conv*offset,0);
	}
}

template<class DataType>
__global__ void convertComplex_kernelf(long long int N, DataType* in, hipFloatComplex* out,
				float conv, DataType offset, int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = make_hipFloatComplex(conv*in[i]-conv*offset,0);
	}
}

template<class DataType, class DataType2>
void convertComplex(long long int N, DataType* in, std::complex<DataType2>* out,
				DataType2 conv, DataType offset, hipStream_t stream){}

template<>
void convertComplex<uint8_t,double>(long long int N, uint8_t* in, std::complex<double>* out,
				double conv, uint8_t offset, hipStream_t stream)
{
	convertComplex_kernel<uint8_t><<<(N/512)+1,512,0,stream>>>(
					N,in,reinterpret_cast<hipDoubleComplex*>(out),conv,offset,512);
	hipDeviceSynchronize();
}

template<>
void convertComplex<uint16_t,double>(long long int N, uint16_t* in, std::complex<double>* out,
				double conv, uint16_t offset, hipStream_t stream)
{
	convertComplex_kernel<uint16_t><<<(N/512)+1,512,0,stream>>>(
					N,in,reinterpret_cast<hipDoubleComplex*>(out),conv,offset,512);
	hipDeviceSynchronize();
}

template<>
void convertComplex<uint8_t,float>(long long int N, uint8_t* in, std::complex<float>* out,
				float conv, uint8_t offset, hipStream_t stream)
{
	convertComplex_kernelf<uint8_t><<<(N/512)+1,512,0,stream>>>(
					N,in,reinterpret_cast<hipFloatComplex*>(out),conv,offset,512);
	hipDeviceSynchronize();
}

template<>
void convertComplex<uint16_t,float>(long long int N, uint16_t* in, std::complex<float>* out,
				float conv, uint16_t offset, hipStream_t stream)
{
	convertComplex_kernelf<uint16_t><<<(N/512)+1,512,0,stream>>>(
					N,in,reinterpret_cast<hipFloatComplex*>(out),conv,offset,512);
	hipDeviceSynchronize();
}

template<class DataType>
__global__ void convert_kernel(long long int N, DataType* in, double* out,
				double conv, DataType offset, int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = conv*in[i]-conv*offset;
	}
}

template<class DataType>
__global__ void convert_kernelf(long long int N, DataType* in, float* out,
				float conv, DataType offset, int threads)
{
	int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = conv*in[i]-conv*offset;
	}
}

template<class DataType, class DataType2>
void convert(long long int N, DataType* in, DataType2* out,
				DataType2 conv, DataType offset, hipStream_t stream){}

template<>
void convert<uint8_t,double>(long long int N, uint8_t* in, double* out,
				double conv, uint8_t offset, hipStream_t stream)
{
	convert_kernel<uint8_t><<<(N/512)+1,512,0,stream>>>(N,in,out,conv,offset,512);
	hipDeviceSynchronize();
}

template<>
void convert<uint16_t,double>(long long int N, uint16_t* in, double* out,
				double conv, uint16_t offset, hipStream_t stream)
{
	convert_kernel<uint16_t><<<(N/512)+1,512,0,stream>>>(N,in,out,conv,offset,512);
	hipDeviceSynchronize();
}

template<>
void convert<uint8_t,float>(long long int N, uint8_t* in, float* out,
				float conv, uint8_t offset, hipStream_t stream)
{
	convert_kernelf<uint8_t><<<(N/512)+1,512,0,stream>>>(N,in,out,conv,offset,512);
	hipDeviceSynchronize();
}

template<>
void convert<uint16_t,float>(long long int N, uint16_t* in, float* out,
				float conv, uint16_t offset, hipStream_t stream)
{
	convert_kernelf<uint16_t><<<(N/512)+1,512,0,stream>>>(N,in,out,conv,offset,512);
	hipDeviceSynchronize();
}

__global__	void autocorrelation_convert_kernel(long long int N, hipFloatComplex* in, 
				double* out, int threads)
{
	// Compute the correlation
	long long int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] = hipCrealf(in[i])*hipCrealf(in[i])+hipCimagf(in[i])*hipCimagf(in[i]);
	}  
}

void autocorrelation_convert(long long int N, std::complex<float>* in, double* out)
{
	int threads = 512;
	long long int blocks = N/threads;
	autocorrelation_convert_kernel<<<blocks+1,threads>>>(N,
					reinterpret_cast<hipFloatComplex*>(in),out,threads);
	hipDeviceSynchronize();
}

__global__ void add_kernel(long long int N, double* in, double* out, int threads)
{
	long long int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] += in[i];
	}  
}

__global__ void add_kernelf(long long int N, float* in, float* out, int threads)
{
	long long int i = threadIdx.x+blockIdx.x*threads;
	if(i<N)
	{
		out[i] += in[i];
	}  
}

template<class DataType>
void add_cuda(long long int N, DataType* in, DataType* out){}

template<>
void add_cuda<double>(long long int N, double* in, double* out)
{
	int threads = 512;
	long long int blocks = N/threads+1;
	add_kernel<<<blocks,threads>>>(N,in,out,threads);
	hipDeviceSynchronize();
}

template<>
void add_cuda<float>(long long int N, float* in, float* out)
{
	int threads = 512;
	long long int blocks = N/threads+1;
	add_kernelf<<<blocks,threads>>>(N,in,out,threads);
	hipDeviceSynchronize();
}
